#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <chrono>
#include <stdexcept>

__global__ void matrixMultiplicationKernel(int* A, int* B, int* C, int N)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < N && col < N)
	{
		int sum = 0;
		for (int i = 0; i < N; i++)
		{
			sum += A[row * N + i] * B[i * N + col];
		}
		C[row * N + col] = sum;
	}
}

void matrixMultiplication(int* A, int* B, int* C, int N)
{
	int* d_A, * d_B, * d_C;
	int size = N * N * sizeof(int);

	hipMalloc(&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	hipMalloc(&d_B, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	hipMalloc(&d_C, size);

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

	matrixMultiplicationKernel << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

void readFromFile(int* A, int* B, int N) {
	std::string line;

	std::ifstream fin(R"(C:\Users\Daniel\source\repos\PPD_CUDA\A.txt)");
	for (int i = 0; i < N; i++) {
		getline(fin, line);
		std::istringstream iss(line);

		for (int j = 0; j < N; j++) {
			iss >> A[i * N + j];
		}
	}
	fin.close();

	std::ifstream fin2(R"(C:\Users\Daniel\source\repos\PPD_CUDA\B.txt)");
	for (int i = 0; i < N; i++) {
		getline(fin2, line);
		std::istringstream iss(line);

		for (int j = 0; j < N; j++) {
			iss >> B[i * N + j];
		}
	}
	fin2.close();
}

void writeResToFile(int* C, int N) {
	std::ofstream fout(R"(C:\Users\Daniel\source\repos\PPD_CUDA\Res.txt)");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			fout << C[i * N + j] << " ";
		}
		fout << std::endl;
	}
	fout.close();
}

int main()
{
	int N = 1000;

	int* A = new int[N * N];
	int* B = new int[N * N];
	int* C = new int[N * N];

	readFromFile(A, B, N);

	auto start = std::chrono::high_resolution_clock::now();

	matrixMultiplication(A, B, C, N);

	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	std::cout << (float)duration / 1000;

	writeResToFile(C, N);

	delete[] A;
	delete[] B;
	delete[] C;

	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}